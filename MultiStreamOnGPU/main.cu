
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

__global__ void KernelA(int *c, int *a, int *b, int N)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < N)
	{
		c[tid] = a[tid] + b[tid];
		tid += blockDim.x * gridDim.x;
	}
	__syncthreads();
}

__global__ void KernelB(int *d, int *a, int *b, int N)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < N)
	{
		d[tid] = a[tid] * b[tid];
		tid += gridDim.x * blockDim.x;
	}
	__syncthreads();
}

int main(int argc, char **argv)
{
	int N;
	N = atoi(argv[1]);
	int *a, *b, *c, *d;
	float elapsedTime;
    hipMallocManaged(&a, N * sizeof(int));
	hipMallocManaged(&b, N * sizeof(int));
	hipMallocManaged(&c, N * sizeof(int));
	hipMallocManaged(&d, N * sizeof(int));
	for (int i = 0; i < N; i++)
	{
		a[i] = i;
		b[i] = i+1;
		c[i] = d[i] = 0;
	}
	hipStream_t stream1, stream2;
	hipStreamCreate(&stream1);
	hipStreamCreate(&stream2);
    hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);
	KernelA<<<3, 8, 0, stream1>>>(c, a, b, N);
	KernelB<<<3, 8, 0, stream2>>>(d, a, b, N);
	hipStreamSynchronize(stream1);
	hipStreamSynchronize(stream2);
    hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	cout << "N: " << N << ", Time taken: " << elapsedTime << "ms" << endl;

	hipFree(a);
	hipFree(b);
	hipFree(c);
	hipFree(d);
    hipStreamDestroy(stream1);
	hipStreamDestroy(stream2);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}



